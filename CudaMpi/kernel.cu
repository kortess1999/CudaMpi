#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>

extern "C" void addWithCuda(int* data, const int arraySize, double* output, const int outputSize, const int blockSize);

__global__ void sumKernel(int* data, const int step, const int blockSize, const int dataSize)
{
    int i = threadIdx.x;
    int blockId = blockIdx.x;

    const int index_first = (2 << step) * i;
    const int index_second = index_first + (1 << step);

    if (index_second < blockSize && blockSize * blockId + index_second < dataSize) {
        data[blockSize * blockId + index_first] = data[blockSize * blockId + index_first] + data[blockSize * blockId + index_second];
    }
}

// Helper function for using CUDA to add vectors in parallel.
extern "C" void addWithCuda(int* data, const int arraySize, double* output, const int outputSize, const int blockSize)
{
    int* d_data = 0;

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&d_data, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(d_data, data, arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    for (int k = 0; k < ceil(log2((double)blockSize)); k++) {
        const int threadCount = ceil(blockSize / pow(2, k));
        sumKernel << <outputSize, threadCount >> > (d_data, k, blockSize, arraySize);

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            goto Error;
        }

        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpy(data, d_data, arraySize * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(data, d_data, arraySize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "sumKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    for (int i = 0; i < outputSize; i++) {
        if (i != outputSize - 1) {
            output[i] = (double)data[blockSize * i] / (double)blockSize;
        }
        else {
            output[i] = (double)data[blockSize * i] / (double)(arraySize - i * blockSize - 1);
        }
    }

Error:
    hipFree(d_data);
}
